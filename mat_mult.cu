#include "hip/hip_runtime.h"
﻿#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "kernel.h"


//Matrix multiplication CUDA GPU function
__global__ void mat_mult_points_kernel(
	double* m,
	double* x, double* y, double* z,
	int numpoints)
{
	double u, v, w;

	//Calculate global thread ID
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	//Boundary check
	if (i < numpoints) {
		u = m[0] * x[i] + m[1] * y[i] + m[2] * z[i] + m[3];
		v = m[4] * x[i] + m[5] * y[i] + m[6] * z[i] + m[7];
		w = m[8] * x[i] + m[9] * y[i] + m[10] * z[i] + m[11];

		x[i] = u;
		y[i] = v;
		z[i] = w;
	}
}

//Host CPU matrix multiplication thread spawner
void mat_mult_device(
	double* X, double* Y, double* Z,
	double m[4][4],
	double* x, double* y, double* z,
	int numpoints)
{
	size_t bytes = numpoints * sizeof(double);
	size_t mat_bytes = 16 * sizeof(double);

	//Allocates memory and copies contents of m onto device
	double* dev_m;
	hipMalloc(&dev_m, mat_bytes);
	hipMemcpy(dev_m, m, mat_bytes, hipMemcpyHostToDevice);

	double* dev_x;
	double* dev_y;
	double* dev_z;

	//Allocate device memory
	hipMalloc(&dev_x, bytes);
	hipMalloc(&dev_y, bytes);
	hipMalloc(&dev_z, bytes);

	//Copy data to device
	hipMemcpy(dev_x, x, bytes, hipMemcpyHostToDevice);
	hipMemcpy(dev_y, y, bytes, hipMemcpyHostToDevice);
	hipMemcpy(dev_z, z, bytes, hipMemcpyHostToDevice);

	//Threads per CTA (1024)
	int NUM_THREADS = 1 << 10;

	//CTAs per grid
	int NUM_BLOCKS = (numpoints + NUM_THREADS - 1) / NUM_THREADS;

	mat_mult_points_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(dev_m, dev_x, dev_y, dev_z, numpoints);

	//Copy data back from device to host
	hipMemcpy(X, dev_x, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(Y, dev_y, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(Z, dev_z, bytes, hipMemcpyDeviceToHost);

	//Free device memory
	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_z);
}
